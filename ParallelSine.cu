#include "hip/hip_runtime.h"
//
// Assignment 1: ParallelSine
// CSCI 415: Networking and Parallel Computation
// Spring 2017
// Name(s): Zhuoran He, Kai Yao 
//
// Sine implementation derived from slides here: http://15418.courses.cs.cmu.edu/spring2016/lecture/basicarch


// standard imports
#include <stdio.h>
#include <math.h>
#include <iomanip>
#include <iostream>
#include <string>
#include <sys/time.h>

// problem size (vector length) N
static const int N = 12345678;

// Number of terms to use when approximating sine
static const int TERMS = 6;

// kernel function (CPU - Do not modify)
void sine_serial(float *input, float *output)
{
  int i;

  for (i=0; i<N; i++) {
      float value = input[i]; 
      float numer = input[i] * input[i] * input[i]; 
      int denom = 6; // 3! 
      int sign = -1; 
      for (int j=1; j<=TERMS;j++) 
      { 
         value += sign * numer / denom; 
         numer *= input[i] * input[i]; 
         denom *= (2*j+2) * (2*j+3); 
         sign *= -1; 
      } 
      output[i] = value; 
    }
}


// kernel function (CUDA device)
// TODO: Implement your graphics kernel here. See assignment instructions for method information

__global__ void sine_parallel(ﬂoat *input, ﬂoat *output)
{
	int i = threadIdx.x               
	
	float value = input[i];
	float numer = value * value * value; 
    int denom = 6; // 3! 
    int sign = -1; 
	for (int j=1; j<=TERMS;j++) 
	{ 
		value += sign * numer / denom; 
		numer *= input[i] * input[i]; 
		denom *= (2*j+2) * (2*j+3); 
		sign *= -1; 
	} 
	output[i] = value;
}

// BEGIN: timing and error checking routines (do not modify)

// Returns the current time in microseconds
long long start_timer() {
	struct timeval tv;
	gettimeofday(&tv, NULL);
	return tv.tv_sec * 1000000 + tv.tv_usec;
}


// Prints the time elapsed since the specified time
long long stop_timer(long long start_time, std::string name) {
	struct timeval tv;
	gettimeofday(&tv, NULL);
	long long end_time = tv.tv_sec * 1000000 + tv.tv_usec;
        std::cout << std::setprecision(5);	
	std::cout << name << ": " << ((float) (end_time - start_time)) / (1000 * 1000) << " sec\n";
	return end_time - start_time;
}

void checkErrors(const char label[])
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}

// END: timing and error checking routines (do not modify)



int main (int argc, char **argv)
{
  //BEGIN: CPU implementation (do not modify)
  float *h_cpu_result = (float*)malloc(N*sizeof(float));
  float *h_input = (float*)malloc(N*sizeof(float));
  //Initialize data on CPU
  int i;
  for (i=0; i<N; i++)
  {
    h_input[i] = 0.1f * i;
  }

  //Execute and time the CPU version
  long long CPU_start_time = start_timer();
  sine_serial(h_input, h_cpu_result);
  long long CPU_time = stop_timer(CPU_start_time, "\nCPU Run Time");
  //END: CPU implementation (do not modify)


  //TODO: Prepare and run your kernel, make sure to copy your results back into h_gpu_result and display your timing results
  float *h_gpu_result = (float*)malloc(N*sizeof(float));
	
	 //declare GPU memory pointers
  float * d_input;
  float * d_output;
  
  //Start timer for entire GPU process
  long long Total_GPU_start_time = start_timer();
  
  //allocate gpu memory
  long long GPU_memAllocate_start_time = start_timer();
  hipMalloc((void **) &d_input, N*sizeof(float));
  hipMalloc((void **) &d_output, N*sizeof(float));
  long long GPU_memAllocate_time = stop_timer(GPU_memAllocate_start_time, "\nGPU Memory Allocation Time");
  
  //transfer the array to the GPU
  long long GPU_memToDevice_start_time = start_timer();
  hipMemcpy(d_input, h_input, N*sizeof(float), cudaMemcpyHostToevice);
  long long GPU_memToDevic_time = stop_timer(GPU_memToDevice_start_time, "\nGPU Copy Memory to Device Time");

  //launch the and time the kernel
  long long GPU_kernel_start_time = start_timer();
  sine_parallel<<<1, >>>(d_output, d_input);
  long long GPU_kernel_time = stop_timer(GPU_kernel_start_time, "\nGPU Kernel Run Time");
  
  //copy back the result to the CPU
  long long GPU_memToHost_start_time = start_timer();
  hipMemcpy(h_gpu_result,d_output, N*sizeof(float), hipMemcpyDeviceToHost);
  long long GPU_memToHost_time = stop_timer(GPU_memToHost_start_time, "\nGPU Copy Memory to Host Time");
  
  //Stop the timer for the entire GPU process
  long long GPU_total_run_time = stop_timer(Total_GPU_start_time, "\nGPU Total Run Time");
  
  //clean memory
  hipFree(d_input);
  hipFree(d_output);

  // Checking to make sure the CPU and GPU results match - Do not modify
  int errorCount = 0;
  for (i=0; i<N; i++)
  {
    if (abs(h_cpu_result[i]-h_gpu_result[i]) > 1e-6)
      errorCount = errorCount + 1;
  }
  if (errorCount > 0)
    printf("Result comparison failed.\n");
  else
    printf("Result comparison passed.\n");

  // Cleaning up memory
  free(h_input);
  free(h_cpu_result);
  free(h_gpu_result);
  return 0;
}






